
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <errno.h>
// #include <malloc.h>

#ifdef _MSC_VER
#define STRERROR(ERRNO, BUF, BUFSIZE) strerror_s((BUF), (BUFSIZE), (ERRNO))
#else
#define STRERROR(ERRNO, BUF, BUFSIZE) strerror_r((ERRNO), (BUF), (BUFSIZE))
#endif


#define MAX_STRING_LENGTH 1000
#define TSIZE 1048576
#define SEED 1159241
#define HASHFN bitwisehash

typedef double real;
typedef struct cooccur_rec {
    int word1;
    int word2;
    real val;
} CREC;
typedef struct hashrec {
    char *word;
    long long num; //count or id
    struct hashrec *next;
} HASHREC;


/* Efficient string comparison */
int scmp( char *s1, char *s2 ) {
    while (*s1 != '\0' && *s1 == *s2) {s1++; s2++;}
    return (*s1 - *s2);
}

/* Simple bitwise hash function */
unsigned int bitwisehash(char *word, int tsize, unsigned int seed) {
    char c;
    unsigned int h;
    h = seed;
    for ( ; (c = *word) != '\0'; word++) h ^= ((h << 5) + c + (h >> 2));
    return (unsigned int)((h & 0x7fffffff) % tsize);
}

/* Create hash table, initialise pointers to NULL */
HASHREC ** inithashtable() {
    int i;
    HASHREC **ht;
    ht = (HASHREC **) malloc( sizeof(HASHREC *) * TSIZE );
    for (i = 0; i < TSIZE; i++) ht[i] = (HASHREC *) NULL;
    return ht;
}

/* Read word from input stream. Return 1 when encounter '\n' or EOF (but separate from word), 0 otherwise.
   Words can be separated by space(s), tab(s), or newline(s). Carriage return characters are just ignored.
   (Okay for Windows, but not for Mac OS 9-. Ignored even if by themselves or in words.)
   A newline is taken as indicating a new document (contexts won't cross newline).
   Argument word array is assumed to be of size MAX_STRING_LENGTH.
   words will be truncated if too long. They are truncated with some care so that they
   cannot truncate in the middle of a utf-8 character, but
   still little to no harm will be done for other encodings like iso-8859-1.
   (This function appears identically copied in vocab_count.c and cooccur.c.)
 */
 int get_word(char *word, FILE *fin) {
    int i = 0, ch;
    for ( ; ; ) {
        ch = fgetc(fin);
        if (ch == '\r') continue;
        if (i == 0 && ((ch == '\n') || (ch == EOF))) {
            word[i] = 0;
            return 1;
        }
        if (i == 0 && ((ch == ' ') || (ch == '\t'))) continue; // skip leading space
        if ((ch == EOF) || (ch == ' ') || (ch == '\t') || (ch == '\n')) {
            if (ch == '\n') ungetc(ch, fin); // return the newline next time as document ender
            break;
        }
        if (i < MAX_STRING_LENGTH - 1)
          word[i++] = ch; // don't allow words to exceed MAX_STRING_LENGTH
    }
    word[i] = 0; //null terminate
    // avoid truncation destroying a multibyte UTF-8 char except if only thing on line (so the i > x tests won't overwrite word[0])
    // see https://en.wikipedia.org/wiki/UTF-8#Description
    if (i == MAX_STRING_LENGTH - 1 && (word[i-1] & 0x80) == 0x80) {
        if ((word[i-1] & 0xC0) == 0xC0) {
            word[i-1] = '\0';
        } else if (i > 2 && (word[i-2] & 0xE0) == 0xE0) {
            word[i-2] = '\0';
        } else if (i > 3 && (word[i-3] & 0xF8) == 0xF0) {
            word[i-3] = '\0';
        }
    }
    return 0;
}

void free_table(HASHREC **ht) {
    int i;
    HASHREC* current;
    HASHREC* tmp;
    for (i = 0; i < TSIZE; i++) {
        current = ht[i];
        while (current != NULL) {
            tmp = current;
            current = current->next;
            free(tmp->word);
            free(tmp);
        }
    }
    free(ht);
}

int find_arg(char *str, int argc, char **argv) {
    int i;
    for (i = 1; i < argc; i++) {
        if (!scmp(str, argv[i])) {
            if (i == argc - 1) {
                printf("No argument given for %s\n", str);
                exit(1);
            }
            return i;
        }
    }
    return -1;
}


void free_fid(FILE **fid, const int num) {
    int i;
    for(i = 0; i < num; i++) {
        if(fid[i] != NULL)
            fclose(fid[i]);
    }
    free(fid);
}


// logs errors when loading files.  call after a failed load
int log_file_loading_error(char *file_description, char *file_name) {
    fprintf(stderr, "Unable to open %s %s.\n", file_description, file_name);
    fprintf(stderr, "Errno: %d\n", errno);
    char error[MAX_STRING_LENGTH];
    STRERROR(errno, error, MAX_STRING_LENGTH);
    fprintf(stderr, "Error description: %s\n", error);
    return errno;
}

typedef struct vocabulary {
    char *word;
    long long count;
} VOCAB;

int verbose = 2; // 0, 1, or 2
long long min_count = 1; // min occurrences for inclusion in vocab
long long max_vocab = 0; // max_vocab = 0 for no limit

/* Vocab frequency comparison; break ties alphabetically */
int CompareVocabTie(const void *a, const void *b) {
    long long c;
    if ( (c = ((VOCAB *) b)->count - ((VOCAB *) a)->count) != 0) return ( c > 0 ? 1 : -1 );
    else return (scmp(((VOCAB *) a)->word,((VOCAB *) b)->word));
    
}

/* Vocab frequency comparison; no tie-breaker */
int CompareVocab(const void *a, const void *b) {
    long long c;
    if ( (c = ((VOCAB *) b)->count - ((VOCAB *) a)->count) != 0) return ( c > 0 ? 1 : -1 );
    else return 0;
}


/* Search hash table for given string, insert if not found */
void hashinsert(HASHREC **ht, char *w) {
    HASHREC     *htmp, *hprv;
    unsigned int hval = HASHFN(w, TSIZE, SEED);
    
    for (hprv = NULL, htmp = ht[hval]; htmp != NULL && scmp(htmp->word, w) != 0; hprv = htmp, htmp = htmp->next);
    if (htmp == NULL) {
        htmp = (HASHREC *) malloc( sizeof(HASHREC) );
        htmp->word = (char *) malloc( strlen(w) + 1 );
        strcpy(htmp->word, w);
        htmp->num = 1;
        htmp->next = NULL;
        if ( hprv==NULL )
            ht[hval] = htmp;
        else
            hprv->next = htmp;
    }
    else {
        /* new records are not moved to front */
        htmp->num++;
        if (hprv != NULL) {
            /* move to front on access */
            hprv->next = htmp->next;
            htmp->next = ht[hval];
            ht[hval] = htmp;
        }
    }
    return;
}


int get_counts() {
    long long i = 0, j = 0, vocab_size = 12500;
    // char format[20];
    char str[MAX_STRING_LENGTH + 1];
    HASHREC **vocab_hash = inithashtable();
    HASHREC *htmp;
    VOCAB *vocab;
    FILE *fid = stdin;
    
    fprintf(stderr, "BUILDING VOCABULARY\n");
    if (verbose > 1) fprintf(stderr, "Processed %lld tokens.", i);
    // sprintf(format,"%%%ds",MAX_STRING_LENGTH);
    while ( ! feof(fid)) {
        // Insert all tokens into hashtable
        int nl = get_word(str, fid);
        if (nl) continue; // just a newline marker or feof
        if (strcmp(str, "<unk>") == 0) {
            fprintf(stderr, "\nError, <unk> vector found in corpus.\nPlease remove <unk>s from your corpus (e.g. cat text8 | sed -e 's/<unk>/<raw_unk>/g' > text8.new)");
            free_table(vocab_hash);
            return 1;
        }
        hashinsert(vocab_hash, str);
        if (((++i)%100000) == 0) if (verbose > 1) fprintf(stderr,"\033[11G%lld tokens.", i);
    }
    if (verbose > 1) fprintf(stderr, "\033[0GProcessed %lld tokens.\n", i);
    vocab = (VOCAB *)malloc(sizeof(VOCAB) * vocab_size);
    for (i = 0; i < TSIZE; i++) { // Migrate vocab to array
        htmp = vocab_hash[i];
        while (htmp != NULL) {
            vocab[j].word = htmp->word;
            vocab[j].count = htmp->num;
            j++;
            if (j>=vocab_size) {
                vocab_size += 2500;
                vocab = (VOCAB *)realloc(vocab, sizeof(VOCAB) * vocab_size);
            }
            htmp = htmp->next;
        }
    }
    if (verbose > 1) fprintf(stderr, "Counted %lld unique words.\n", j);
    if (max_vocab > 0 && max_vocab < j)
        // If the vocabulary exceeds limit, first sort full vocab by frequency without alphabetical tie-breaks.
        // This results in pseudo-random ordering for words with same frequency, so that when truncated, the words span whole alphabet
        qsort(vocab, j, sizeof(VOCAB), CompareVocab);
    else max_vocab = j;
    qsort(vocab, max_vocab, sizeof(VOCAB), CompareVocabTie); //After (possibly) truncating, sort (possibly again), breaking ties alphabetically
    
    for (i = 0; i < max_vocab; i++) {
        if (vocab[i].count < min_count) { // If a minimum frequency cutoff exists, truncate vocabulary
            if (verbose > 0) fprintf(stderr, "Truncating vocabulary at min count %lld.\n",min_count);
            break;
        }
        printf("%s %lld\n",vocab[i].word,vocab[i].count);
    }
    
    if (i == max_vocab && max_vocab < j) if (verbose > 0) fprintf(stderr, "Truncating vocabulary at size %lld.\n", max_vocab);
    fprintf(stderr, "Using vocabulary of size %lld.\n\n", i);
    free_table(vocab_hash);
    free(vocab);
    return 0;
}

int main(int argc, char **argv) {
    if (argc == 2 &&
        (!scmp(argv[1], "-h") || !scmp(argv[1], "-help") || !scmp(argv[1], "--help"))) {
        printf("Simple tool to extract unigram counts\n");
        printf("Author: Jeffrey Pennington (jpennin@stanford.edu)\n\n");
        printf("Usage options:\n");
        printf("\t-verbose <int>\n");
        printf("\t\tSet verbosity: 0, 1, or 2 (default)\n");
        printf("\t-max-vocab <int>\n");
        printf("\t\tUpper bound on vocabulary size, i.e. keep the <int> most frequent words. The minimum frequency words are randomly sampled so as to obtain an even distribution over the alphabet.\n");
        printf("\t-min-count <int>\n");
        printf("\t\tLower limit such that words which occur fewer than <int> times are discarded.\n");
        printf("\nExample usage:\n");
        printf("./vocab_count -verbose 2 -max-vocab 100000 -min-count 10 < corpus.txt > vocab.txt\n");
        return 0;
    }

    int i;
    if ((i = find_arg((char *)"-verbose", argc, argv)) > 0) verbose = atoi(argv[i + 1]);
    if ((i = find_arg((char *)"-max-vocab", argc, argv)) > 0) max_vocab = atoll(argv[i + 1]);
    if ((i = find_arg((char *)"-min-count", argc, argv)) > 0) min_count = atoll(argv[i + 1]);
    return get_counts();
}


